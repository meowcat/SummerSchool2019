#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "util.hpp"

// host implementation of dot product
double dot_host(const double *x, const double* y, int n) {
    double sum = 0;
    for(auto i=0; i<n; ++i) {
        sum += x[i]*y[i];
    }
    return sum;
}

template <int THREADS>
__global__
void dot_gpu_kernel(const double *x, const double* y, double *result, int n) {
    __shared__ double buf[THREADS];
    int i = threadIdx.x;

    // initialize this thread's contribution to the reduction
    buf[i] = 0;
    if (i<n) buf[i] = x[i]*y[i];

    // perform binary reduction
    int mid = THREADS/2;
    while (mid) {
        __syncthreads();
        if (i<mid)
            buf[i] += buf[i+mid];
        mid /= 2;
    }

    // the first thread writes the result
    // no need to __syncthreads()
    if (i==0)
        *result = buf[0];
}

template <int THREADS>
__global__
void dot_gpu_kernel_full(const double *x, const double* y, double *result, int n) {
    __shared__ double buf[THREADS];
    int lid = threadIdx.x;
    int i = threadIdx.x+blockIdx.x*blockDim.x;

    buf[lid] = i<n? x[i]*y[i]: 0;

    int mid = THREADS/2;

    while (mid) {
        __syncthreads();
        if (lid<mid)
            buf[lid] += buf[lid+mid];
        mid /= 2;
    }

    if (lid==0) atomicAdd(result, buf[0]);
}

double dot_gpu(const double *x, const double* y, int n) {
    static double* result = malloc_managed<double>(1);
    *result = 0.;
    //dot_gpu_kernel<1024><<<1, 1024>>>(x, y, result, n);
    dot_gpu_kernel_full<1024><<<(n+1023)/1024, 1024>>>(x, y, result, n);
    hipDeviceSynchronize();
    return *result;
}

int main(int argc, char** argv) {
    size_t n = read_arg(argc, argv, 1, 4);

    auto size_in_bytes = n * sizeof(double);

    std::cout << "dot product CUDA of length n = " << n
              << " : " << size_in_bytes*1e-9 << "MB\n";

    auto x_h = malloc_host<double>(n, 2.);
    auto y_h = malloc_host<double>(n);
    for(auto i=0; i<n; ++i) {
        y_h[i] = rand()%10;
    }

    auto x_d = malloc_device<double>(n);
    auto y_d = malloc_device<double>(n);

    // copy initial conditions to device
    copy_to_device<double>(x_h, x_d, n);
    copy_to_device<double>(y_h, y_d, n);

    auto result   = dot_gpu(x_d, y_d, n);
    auto expected = dot_host(x_h, y_h, n);
    printf("expected %f got %f\n", (float)expected, (float)result);

    return 0;
}

