#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>

#include "util.hpp"

// TODO : implement a kernel that reverses a string of length n in place
// void reverse_string(char* str, int n)
__global__
void reverse_string(char* str, int n) {
    __shared__ char buffer[1024];
    int i = threadIdx.x;
    if (i<n) {
        buffer[i] = str[i];
        __syncthreads();
        str[i] = buffer[n-i-1];
    }
}

__global__
void reverse_string_nosmem(char* str, int n) {
    int i = threadIdx.x;
    if (i<n) {
        char c = str[n-i-1];
        __syncthreads();
        str[i] = c;
    }
}

int main(int argc, char** argv) {
    // check that the user has passed a string to reverse
    if(argc<2) {
        std::cout << "useage : ./string_reverse \"string to reverse\"\n" << std::endl;
        exit(0);
    }

    // determine the length of the string, and copy in to buffer
    auto n = strlen(argv[1]);
    auto string = malloc_managed<char>(n+1);
    std::copy(argv[1], argv[1]+n, string);
    string[n] = 0; // add null terminator

    std::cout << "string to reverse:\n" << string << "\n";

    // TODO : call the string reverse function
    //reverse_string<<<1, 1024>>>(string, n);
    reverse_string_nosmem<<<1, 1024>>>(string, n);

    // print reversed string
    hipDeviceSynchronize();
    std::cout << "reversed string:\n" << string << "\n";

    // free memory
    hipFree(string);

    return 0;
}

